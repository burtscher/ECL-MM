#include "hip/hip_runtime.h"
/*
ECL-MM: This code computes a maximum matching in a bipartite graph.

Copyright (c) 2025, Anju Mongandampulath Akathoott and Martin Burtscher

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://cs.txstate.edu/~burtscher/research/ECL-MM/ and at https://github.com/burtscher/ECL-MM.

Publication: This work is described in detail in the following paper.
Anju Mongandampulath Akathoott and Martin Burtscher. "A Bidirectional GPU Algorithm for Computing Maximum Matchings in Bipartite Graphs." Proceedings of the 39th IEEE International Parallel and Distributed Processing Symposium. June 2025.
*/


#include <cstdlib>
#include <cstdio>
#include <string>
#include <time.h>
#include <algorithm>
#include <climits>
#include <cassert>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <sstream>
#include "ECLatomic.h"
#include "ECLgraph.h"

static const int Device = 0;
static const int TPB = 256; // Threads per block
static const int NONE = -1; 

static const int threshold = 16;

static bool goAgain, apFound;
static int sizeOfA, sizeOfB, numEdges, apSearch_itrCount, totalNumPaths, totalPathLengths, wlSize, nextWlSize, startIndexOfB, numPathsAugmentedInCurItr, n; // n = sizeOfA + sizeOfB
static int *mate;

static int *d_nbrIndx, *d_edge, *d_mate, *d_nextOption, *d_itrFlag, *d_endPoint, *d_parent, *d_source, *d_workList, *d_nextWorkList;
static int *d_wlSize, *d_nextWlSize, *d_totalPathLengths, *d_totalNumPaths;
static bool *d_apFound, *d_goAgain;

struct CPUTimer
{
  timeval beg, end;
  CPUTimer() {}
  ~CPUTimer() {}
  void start() {gettimeofday(&beg, NULL);}
  double elapsed() {gettimeofday(&end, NULL); return end.tv_sec - beg.tv_sec + (end.tv_usec - beg.tv_usec) / 1000000.0;}
};


static void printGraphSpec(char* fileName)
{
  std::string shortName(fileName);
  std::istringstream iss(shortName);
  std::string s;
  getline(iss, s, '.');
  std::istringstream iss2(s);
  std::string token;
  while (getline(iss2, token, '/'));
  printf("input=%s\n", token.c_str());
  printf("|A| = %d\n|B| = %d\n|E| = %d\n", sizeOfA, sizeOfB, numEdges);
}


static __global__ void populateWL_kernel(const int* const __restrict__ d_mate, const int* const __restrict__ d_nbrIndx, int* __restrict__ d_parent, int* __restrict__ d_source, int* __restrict__ d_workList, int n, int* __restrict__ d_wlSize)
{
  int v = blockDim.x * blockIdx.x + threadIdx.x;
  if (v < n) {
    if (d_mate[v] == NONE && (d_nbrIndx[v + 1] != d_nbrIndx[v])) {
      int index = atomicAdd(d_wlSize, 1);
      d_workList[index] = v;
      d_parent[v] = v;
      d_source[v] = v;
    }
  }
}


static __global__ void recreateFrontier(int* __restrict__ d_workList, int* __restrict__ d_nextWorkList, int* __restrict__ d_parent, int* __restrict__ d_source, const int* const __restrict__ d_endPoint, int* __restrict__ d_wlSize, int* __restrict__ d_nextWlSize, const int n, const int sizeOfA, const int startIndexOfB)
{
  int v = blockDim.x * blockIdx.x + threadIdx.x; 
  if ((v < n) && (d_source[v] != NONE)) {
    if (d_endPoint[d_source[v]] == NONE) { // v is an active vertex
      if ((v < sizeOfA && d_source[v] < sizeOfA) || (v >= startIndexOfB && d_source[v] >= startIndexOfB)) {
        int index = atomicAdd(d_wlSize, 1);
        d_workList[index] = v;
      } else {
        int index = atomicAdd(d_nextWlSize, 1);
        d_nextWorkList[index] = v;
      }
    } else {
      // v is a vertex in a dead tree
      d_parent[v] = NONE;
      d_source[v] = NONE;
    }
  } 
}


static void reuse()
{
  wlSize = 0;
  nextWlSize = 0;
  hipMemcpyAsync(d_wlSize, &wlSize, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_nextWlSize, &nextWlSize, sizeof(int), hipMemcpyHostToDevice);
  recreateFrontier<<<(n + TPB - 1) / TPB, TPB>>>(d_workList, d_nextWorkList, d_parent, d_source, d_endPoint, d_wlSize, d_nextWlSize, n, sizeOfA, startIndexOfB);
  hipMemcpyAsync(&wlSize, d_wlSize, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&nextWlSize, d_nextWlSize, sizeof(int), hipMemcpyDeviceToHost);
}


static void setUp()
{
  if (numPathsAugmentedInCurItr > 0) {
    reuse();
  }
  else {
    hipMemset(d_endPoint, NONE, n * sizeof(int));
    hipMemset(d_parent, NONE, n * sizeof(int));
    hipMemset(d_source, NONE, n * sizeof(int));
    wlSize = 0;
    nextWlSize = 0;
    hipMemcpyAsync(d_wlSize, &wlSize, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_nextWlSize, &nextWlSize, sizeof(int), hipMemcpyHostToDevice);
    populateWL_kernel<<<(n + TPB - 1) / TPB, TPB>>>(d_mate, d_nbrIndx, d_parent, d_source, d_workList, n, d_wlSize);
    hipMemcpy(&wlSize, d_wlSize, sizeof(int), hipMemcpyDeviceToHost);
  }
}


static __global__ void levelOne_kernel(const int* const __restrict__ d_wlSize, const int* const __restrict__ d_workList, int* __restrict__ d_source, int* __restrict__ d_endPoint, const int* const __restrict__ d_nbrIndx, const int* const __restrict__ d_edge, const int* const __restrict__ d_mate, int* __restrict__ d_parent, int* __restrict__ d_nextWlSize, int* __restrict__ d_nextWorkList, const int sizeOfA, const int startIndexOfB, bool* __restrict__ d_apFound)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (__any_sync(~0, i < (*d_wlSize))) {
    int beg, end, v, src_v, mateOf_v;
    int deg = -1;
    if (i < (*d_wlSize)) { 
      v = d_workList[i];
      src_v = atomicRead(&d_source[v]);
      mateOf_v = d_mate[v];
      if (atomicRead(&d_endPoint[src_v]) == NONE) {
        beg = d_nbrIndx[v];
        end = d_nbrIndx[v + 1];
        deg = end - beg;
        if (deg < threshold) { 
          // one thread does work:
          for (int j = beg; j < end; j++) {
            int nbr = d_edge[j];
            if (mateOf_v != nbr) { // (v, nbr) must be an unmatched edge
              int src_nbr = atomicRead(&d_source[nbr]);
              if (src_nbr != NONE) { // potential middle edge of an AP
                if ((src_v < sizeOfA && src_nbr >= startIndexOfB) || (src_v >= startIndexOfB && src_nbr < sizeOfA)) {
                  int v1, v2, src_v1, src_v2;
                  if (src_v < sizeOfA) {
                    v1 = v;
                    v2 = nbr;
                    src_v1 = src_v;
                    src_v2 = src_nbr;
                  } else {
                    v1 = nbr;
                    v2 = v;
                    src_v1 = src_nbr;
                    src_v2 = src_v;
                  }
                  int valToSet1;
                  if (d_mate[v1] != NONE)
                    valToSet1 = d_mate[v1];
                  else
                    valToSet1 = v1;
                  if ((atomicCAS(&(d_endPoint[src_v1]), NONE, valToSet1)) == NONE) { // indicates firstUpdateSuccess
                    int valToSet2;
                    if (d_mate[v2] != NONE)
                      valToSet2 = d_mate[v2];
                    else
                      valToSet2 = v2;

                    if ((atomicCAS(&(d_endPoint[src_v2]), NONE, valToSet2)) == NONE) { // indicates secUpdatesuccess
                      if (d_parent[v1] != v1) {
                        d_parent[v1] = v2;
                        atomicWrite(&d_endPoint[src_v2], v1);
                      } else {
                        d_parent[v2] = v1;
                        atomicWrite(&d_endPoint[src_v1], v2);
                      }
                      atomicWrite(d_apFound, true);
                      break;
                    } else {
                      atomicWrite(&d_endPoint[src_v1], NONE); // Resetting the first update, as this thread could not write to the second location successfully
                    }
                  }
                }
              } else {
                if ((atomicCAS(&(d_parent[nbr]), NONE, v)) == NONE) { // success
                  atomicWrite(&d_source[nbr], src_v);
                  int index = atomicAdd(d_nextWlSize, 1);
                  d_nextWorkList[index] = nbr;
                }
              }
            }
          }
        } 
      } 
    }
    // Work by threads with deg >= threshold:
    const int WS = 32; //warp size
    const int lane = threadIdx.x % WS;
    int bal = __ballot_sync(~0, deg >= threshold);
    while (bal != 0) {
      const int who = __ffs(bal) - 1;
      bal &= bal - 1;
      const int wbeg = __shfl_sync(~0, beg, who);
      const int wend = __shfl_sync(~0, end, who);
      const int wv = __shfl_sync(~0, v, who);
      const int wmateOf_v = __shfl_sync(~0, mateOf_v, who);
      const int wsrc_v = __shfl_sync(~0, src_v, who);
      for (int j = wbeg + lane; j < wend; j += WS) {
        int nbr = d_edge[j];
        if (wmateOf_v != nbr) { // (wv, nbr) must be an unmatched edge
          int src_nbr = atomicRead(&d_source[nbr]);
          if (src_nbr != NONE) { // potential middle edge of an AP
            if ((wsrc_v < sizeOfA && src_nbr >= startIndexOfB) || (wsrc_v >= startIndexOfB && src_nbr < sizeOfA)) {
              int v1, v2, src_v1, src_v2;
              if (wsrc_v < sizeOfA) {
                v1 = wv;
                v2 = nbr;
                src_v1 = wsrc_v;
                src_v2 = src_nbr;
              } else {
                v1 = nbr;
                v2 = wv;
                src_v1 = src_nbr;
                src_v2 = wsrc_v;
              }
              int valToSet1;
              if (d_mate[v1] != NONE)
                valToSet1 = d_mate[v1];
              else
                valToSet1 = v1;
              if ((atomicCAS(&(d_endPoint[src_v1]), NONE, valToSet1)) == NONE) { // indicates firstUpdateSuccess
                int valToSet2;
                if (d_mate[v2] != NONE)
                  valToSet2 = d_mate[v2];
                else
                  valToSet2 = v2;

                if ((atomicCAS(&(d_endPoint[src_v2]), NONE, valToSet2)) == NONE) { // indicates secUpdatesuccess
                  if (d_parent[v1] != v1) {
                    d_parent[v1] = v2;
                    atomicWrite(&d_endPoint[src_v2], v1);
                  } else {
                    d_parent[v2] = v1;
                    atomicWrite(&d_endPoint[src_v1], v2);
                  }
                  atomicWrite(d_apFound, true);
                  break;
                } else {
                  atomicWrite(&d_endPoint[src_v1], NONE); // Resetting the first update, as this thread could not write to the second location successfully
                }
              }
            }
          } else {
            if ((atomicCAS(&(d_parent[nbr]), NONE, wv)) == NONE) { // success
              atomicWrite(&d_source[nbr], wsrc_v);
              int index = atomicAdd(d_nextWlSize, 1);
              d_nextWorkList[index] = nbr;
            }
          }
        }
      }
    }
  }
}


static __global__ void levelTwo_kernel(const int* const __restrict__ d_nextWlSize, const int* const __restrict__ d_nextWorkList, const int* const __restrict__ d_mate, int* __restrict__ d_source, int* __restrict__ d_parent, int* __restrict__ d_endPoint, int* __restrict__ d_wlSize, int* __restrict__ d_workList, bool* __restrict__ d_goAgain, bool* __restrict__ d_apFound, int* __restrict__ d_totalPathLengths, const int sizeOfA, const int startIndexOfB)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < (*d_nextWlSize)) {
    int v = d_nextWorkList[i];
    int nbr = d_mate[v];
    if (d_source[nbr] != NONE) {
      if ((d_source[v] < sizeOfA && d_source[nbr] >= startIndexOfB) || (d_source[v] >= startIndexOfB && d_source[nbr] < sizeOfA)) {
        // Middle edge. Process only in one direction.
        int v1, v2;
        if (d_source[v] < sizeOfA) {
          v1 = v;
          v2 = nbr;
        } else {
          v1 = nbr;
          v2 = v;
        }
        if (atomicCAS(&(d_endPoint[d_source[v1]]), NONE, v1) == NONE) { // indicates firstUpdateSuccess
          if (atomicCAS(&(d_endPoint[d_source[v2]]), NONE, v2) == NONE) {
            atomicWrite(d_apFound, true);
            atomicAdd(d_totalPathLengths, 1);
          } else {
            d_endPoint[d_source[v1]] = NONE; // Resetting the first update, as this thread could not write to the second location successfully
          }
        }
      }
    } else {
      d_parent[nbr] = v;
      d_source[nbr] = d_source[v];
      atomicWrite(d_goAgain, true);
      int index = atomicAdd(d_wlSize, 1);
      d_workList[index] = nbr;
    }
  }
}


static void searchForAP()
{
  setUp();
  goAgain = true;
  while (goAgain) {
    goAgain = false;
    hipMemcpyAsync(d_goAgain, &goAgain, sizeof(bool), hipMemcpyHostToDevice);
    if (wlSize > 0) {
      levelOne_kernel<<<(wlSize + TPB - 1) / TPB, TPB>>>(d_wlSize, d_workList, d_source, d_endPoint, d_nbrIndx, d_edge, d_mate, d_parent, d_nextWlSize, d_nextWorkList, sizeOfA, startIndexOfB, d_apFound);
    }
    wlSize = 0;
    hipMemcpyAsync(d_wlSize, &wlSize, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(&nextWlSize, d_nextWlSize, sizeof(int), hipMemcpyDeviceToHost);
    if (nextWlSize > 0) {
      levelTwo_kernel<<<(nextWlSize + TPB - 1) / TPB, TPB>>>(d_nextWlSize, d_nextWorkList, d_mate, d_source, d_parent, d_endPoint, d_wlSize, d_workList, d_goAgain, d_apFound, d_totalPathLengths, sizeOfA, startIndexOfB);
    }

    hipMemcpyAsync(&goAgain, d_goAgain, sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(&wlSize, d_wlSize, sizeof(int), hipMemcpyDeviceToHost); 
    if (goAgain) {
      nextWlSize = 0;
      hipMemcpyAsync(d_nextWlSize, &nextWlSize, sizeof(int), hipMemcpyHostToDevice);
    }
  }
  hipMemcpy(&apFound, d_apFound, sizeof(bool), hipMemcpyDeviceToHost);
}


static __global__ void augFromMidToFreevertex(const int* const __restrict__ d_source, const int* const __restrict__ d_parent, const int* const __restrict__ d_endPoint, int* __restrict__ d_mate, const int sizeOfA, int* __restrict__ d_totalNumPaths, int* __restrict__ d_totalPathLengths, const int n)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    if (d_source[i] == i) {
      int startVertex = d_endPoint[i];
      if (startVertex != -1) {
        if (d_parent[startVertex] == startVertex) {
          // Nothing to do towards free end. Trivial, as it is the end point.
          if (startVertex < sizeOfA) { // Inc only for one of the 2 halfpaths in an AP
            atomicAdd(d_totalNumPaths, 1);
          }
        } else {
          if (startVertex < sizeOfA) {
            atomicAdd(d_totalPathLengths, 1); // To count the middle edge
            atomicAdd(d_totalNumPaths, 1);
          }
          int v1 = startVertex;
          int v2 = d_parent[v1];
          while (true) {
            d_mate[v1] = v2;
            d_mate[v2] = v1;
            atomicAdd(d_totalPathLengths, 1);
            if (d_parent[v2] == v2) {
              break;
            } else {
              v1 = d_parent[v2];
              v2 = d_parent[v1];
              atomicAdd(d_totalPathLengths, 1);
            }
          }
        }
      }
    }
  }
}


static void augment()
{
  int oldNumPaths = totalNumPaths;
  augFromMidToFreevertex<<<(n + TPB - 1) / TPB, TPB>>>(d_source, d_parent, d_endPoint, d_mate, sizeOfA, d_totalNumPaths, d_totalPathLengths, n);
  hipMemcpyAsync(&totalNumPaths, d_totalNumPaths, sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  numPathsAugmentedInCurItr = totalNumPaths - oldNumPaths;
}


static int findSizeOfMatching()
{
  int numMatchedEdges = 0;
  for (int a = 0; a < sizeOfA; a++) {
    if (mate[a] != -1)
      numMatchedEdges++;
  }
  return numMatchedEdges;
}


static void freeMemory()
{
  delete[] mate;
  hipFree(d_nextOption);
  hipFree(d_itrFlag);
  hipFree(d_nbrIndx);
  hipFree(d_edge);
  hipFree(d_mate);
  hipFree(d_endPoint);
  hipFree(d_parent);
  hipFree(d_source);
  hipFree(d_workList);
  hipFree(d_nextWorkList);
  hipFree(d_wlSize);
  hipFree(d_nextWlSize);
  hipFree(d_goAgain);
  hipFree(d_apFound);
  hipFree(d_totalPathLengths);
  hipFree(d_totalNumPaths);
}


static __device__ bool hasHigherPrio(const int a, const int a1, const int deg_a, int* __restrict__ d_nextOption, const int* const __restrict__ d_nbrIndx)
{
  if (a1 == NONE)
    return true;
  else {
    int nextOption_a1 = atomicRead(&d_nextOption[a1]); // atomic read since a1 may write to this location in parallel
    int nextOp_a = d_nextOption[a]; // Only a can write to this location
    int deg_a1 = d_nbrIndx[a1 + 1] - d_nbrIndx[a1];
    return ((deg_a - nextOp_a < deg_a1 - nextOption_a1) || ((deg_a - nextOp_a == deg_a1 - nextOption_a1) && a < a1)); // checking if a has fewer options left compared to a1
  }
}


static __global__ void init_processANodes(const int itr, const int size, bool* __restrict__ d_repeat, int* __restrict__ d_mate, const int* const __restrict__ d_nbrIndx, const int* const __restrict__ d_edge, int* __restrict__ d_nextOption, int* __restrict__ d_itrFlag)
{
  int a = blockDim.x * blockIdx.x + threadIdx.x;
  if (a < size) {
    if (atomicRead(&d_mate[a]) == NONE) {
      for (int i = d_nbrIndx[a] + d_nextOption[a]; i < d_nbrIndx[a + 1]; i++) {
        atomicAdd(&d_nextOption[a], 1);
        int myDeg = d_nbrIndx[a + 1] - d_nbrIndx[a];
        int b = d_edge[i];
        int curMateOfb;
        curMateOfb = atomicRead(&d_mate[b]);
        bool success = false;
        while (hasHigherPrio(a, curMateOfb, myDeg, d_nextOption, d_nbrIndx)) {
          int prevVal = atomicCAS(&(d_mate[b]), curMateOfb, a);
          if (prevVal == curMateOfb) {
            success = true;
            break;
          } else {
            curMateOfb = prevVal;
          }
        }
        if (success) {
          atomicWrite(&d_itrFlag[b], itr);
          if (curMateOfb != NONE) {
            atomicWrite(&d_mate[curMateOfb], NONE);
            atomicWrite(d_repeat, true);
          }
          break;
        }
      }
    }
  }
}


static __global__ void init_processBNodes(const int itr, const int sizeOfB, const int startIndexOfB, const int* const __restrict__ d_itrFlag, int* __restrict__ d_mate)
{
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < sizeOfB) {
    int b = startIndexOfB + id;
    if (d_itrFlag[b] == itr) {
      int a = d_mate[b];
      d_mate[a] = b;
    }
  }
}


static void d_init()
{
  hipMemset(d_nextOption, 0, sizeOfA * sizeof(int));
  hipMemset(d_itrFlag, 0, n * sizeof(int));
  int itr = 0;
  bool repeat;
  bool *d_repeat;
  hipMalloc((void **)&d_repeat, sizeof(bool));
  do {
    repeat = false;
    hipMemcpy(d_repeat, &repeat, sizeof(bool), hipMemcpyHostToDevice);
    itr++;
    init_processANodes<<<(sizeOfA + TPB - 1) / TPB, TPB>>>(itr, sizeOfA, d_repeat, d_mate, d_nbrIndx, d_edge, d_nextOption, d_itrFlag);
    init_processBNodes<<<(sizeOfB + TPB - 1) / TPB, TPB>>>(itr, sizeOfB, startIndexOfB, d_itrFlag, d_mate);
    hipMemcpy(&repeat, d_repeat, sizeof(bool), hipMemcpyDeviceToHost);
  } while (repeat);
  printf("initIterations = %d\n", itr);
  hipFree(d_repeat);
}


static void allocateAndInitDS(ECLgraph& g)
{
  mate = new int [n];
  // Arrays:
  hipMalloc((void **)&d_nextOption, sizeOfA * sizeof(int));
  hipMalloc((void **)&d_itrFlag, n * sizeof(int));
  hipMalloc((void **)&d_nbrIndx, (n + 1) * sizeof(int));
  hipMalloc((void **)&d_edge, numEdges * sizeof(int));
  hipMalloc((void **)&d_mate, n * sizeof(int));
  hipMalloc((void **)&d_endPoint, n * sizeof(int));
  hipMalloc((void **)&d_parent, n * sizeof(int));
  hipMalloc((void **)&d_source, n * sizeof(int));
  hipMalloc((void **)&d_workList, n * sizeof(int));
  hipMalloc((void **)&d_nextWorkList, n * sizeof(int));

  // Variables:
  hipMalloc((void **)&d_wlSize, sizeof(int));
  hipMalloc((void **)&d_nextWlSize, sizeof(int));
  hipMalloc((void **)&d_goAgain, sizeof(bool));
  hipMalloc((void **)&d_apFound, sizeof(bool));
  hipMalloc((void **)&d_totalPathLengths, sizeof(int));
  hipMalloc((void **)&d_totalNumPaths, sizeof(int));
  
  hipMemcpyAsync(d_totalPathLengths, &totalPathLengths, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_totalNumPaths, &totalNumPaths, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_nbrIndx, g.nindex, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_edge, g.nlist, numEdges * sizeof(int), hipMemcpyHostToDevice);
  hipMemset(d_mate, NONE, n * sizeof(int));
}


static void printDegreeDetails(ECLgraph& g)
{
  int* degree = new int [g.nodes];
  int sum = 0;
  for (int v = 0; v < g.nodes; v++) {
    degree[v] = g.nindex[v + 1] - g.nindex[v];
    sum += degree[v];
  }
  int minDeg = *(std::min_element(degree, (degree + g.nodes)));
  int maxDeg = *(std::max_element(degree, (degree + g.nodes)));
  assert(sum >= 0 && sum < INT_MAX);
  float avgDeg = (float)sum / g.nodes;
  printf("minDeg = %d\nmaxDeg = %d\navgDeg = %.2f\n", minDeg, maxDeg, avgDeg);
  delete[] degree;
}


int main(int argc, char* argv[])
{
  printf("ECL-MM v1.0\n\n");  fflush(stdout);

  if (argc < 2) {
    fprintf(stderr, "USAGE: %s <inputFileName(s)>\nExiting...\n", argv[0]);
    exit(-1);
  }
  printf("threadsPerBlock = %d\n", TPB);
  // Processing one input
  CPUTimer readTimer;
  readTimer.start();
  ECLgraph g = readECLgraph(argv[1]);
  double rt = readTimer.elapsed();
  printf("GraphReadTime = %.2f s\n", rt);

  printf("threshold for enabling warp-centric processing= %d\n", threshold);
  
  n = g.nodes;
  sizeOfA = n / 2;
  sizeOfB = sizeOfA;
  startIndexOfB = sizeOfA;
  numEdges = g.edges;
  printGraphSpec(argv[1]);
  printDegreeDetails(g);
  totalNumPaths = 0;
  totalPathLengths = 0;
  numPathsAugmentedInCurItr = 0;

  // check GPU
  hipSetDevice(Device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, Device);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {fprintf(stderr, "ERROR: no CUDA capable device detected\n\n"); exit(-1);}
  const int SMs = deviceProp.multiProcessorCount;
  const int mTpSM = deviceProp.maxThreadsPerMultiProcessor;
  printf("GPU: %s with %d SMs and %d mTpSM (%.1f MHz and %.1f MHz)\n", deviceProp.name, SMs, mTpSM, deviceProp.clockRate * 0.001, deviceProp.memoryClockRate * 0.001);
  const float bw = 2.0 * deviceProp.memoryClockRate * (deviceProp.memoryBusWidth / 8) * 0.000001;
  printf("     %.1f GB/s (%.1f+%.1f) peak bandwidth (%d-bit bus)\n\n", bw, bw / 2, bw / 2, deviceProp.memoryBusWidth);
  CheckCuda(__LINE__);

  allocateAndInitDS(g);

  // Initialization of matching:
  CPUTimer t1;
  t1.start();
  d_init();
  double initTime = t1.elapsed();
  hipMemcpy(mate, d_mate, n* sizeof(int), hipMemcpyDeviceToHost);
  printf("initRuntime = %f\n", initTime);
  int m = findSizeOfMatching();
  printf("initialM = %d\n", m);
  printf("initialM_AsPercent = %.2f%%\n", 100.0 * m / sizeOfA);

  // AP Search Phase:
  apSearch_itrCount = 0;
  double apAndAugTime = 0.0;
  CPUTimer ts;
  ts.start();
  do {
    apFound = false;
    hipMemcpyAsync(d_apFound, &apFound, sizeof(bool), hipMemcpyHostToDevice);
    apSearch_itrCount++;
    searchForAP();
    if (apFound) {
      augment();
    } else {
      apAndAugTime = ts.elapsed();
      printf("apSearchTime = %f\n", apAndAugTime);
      printf("totalRunTime = %f\n", initTime + apAndAugTime );
      printf("apSearchItr = %d\n", apSearch_itrCount);
    }
  } while (apFound);

  hipMemcpy(mate, d_mate, n * sizeof(int), hipMemcpyDeviceToHost);
  int finalM = findSizeOfMatching();
  hipMemcpy(&totalNumPaths, d_totalNumPaths, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&totalPathLengths, d_totalPathLengths, sizeof(int), hipMemcpyDeviceToHost);

  printf("finalM = %d\n", finalM);
  printf("finalM_AsPercent = %.2f%%\n", 100.0 * finalM / sizeOfA);
  printf("avgAPLen = %f\n", (float)totalPathLengths / totalNumPaths);
  printf("numNodesDivByTotalRunTime = %lf\n", (double)n / (initTime + apAndAugTime));
  printf("numNodesDivByAPSearchPhaseTime = %lf\n", (double)n / (apAndAugTime));
  printf("numEdgesDivByTotalRunTime = %lf\n", (double)numEdges / (initTime + apAndAugTime));
  printf("numEdgesDivByAPSearchPhaseTime = %lf\n", (double)numEdges / (apAndAugTime));
  printf("-----------------------------\n");
  freeECLgraph(g);
  freeMemory();
}
